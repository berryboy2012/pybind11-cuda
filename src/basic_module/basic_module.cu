#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
namespace py=pybind11;
template <typename T>
__global__ void kernel
(T *vec, T scalar, size_t num_elements)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    vec[idx] = vec[idx] * scalar;
  }
}

template <typename T>
void run_kernel
(T *vec, T scalar, size_t num_elements)
{
  dim3 dimBlock(256, 1, 1);
  dim3 dimGrid((unsigned int)ceil((double)num_elements / dimBlock.x));
  
  kernel<T><<<dimGrid, dimBlock>>>
    (vec, scalar, num_elements);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream strstr;
    strstr << "run_kernel launch failed" << std::endl;
    strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
    strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
    strstr << hipGetErrorString(error);
    throw std::runtime_error(strstr.str());
  }
}

template <typename Tv>
void map_array(py::array_t<Tv, py::array::c_style | py::array::forcecast> &vec,
               const py::buffer& scalar)
{
  auto vec_info = vec.request(true);
  auto scalar_info = scalar.request();
  auto sca = *static_cast<Tv *>(scalar_info.ptr);

  if (vec_info.ndim != 1) {
    std::stringstream strstr;
    strstr << "vec.ndim != 1" << std::endl;
    strstr << "vec.ndim: " << vec_info.ndim << std::endl;
    throw std::runtime_error(strstr.str());
  }

  auto size = vec_info.shape[0];
  auto size_bytes = size*sizeof(Tv);
  Tv *gpu_ptr;
  hipError_t error = hipMalloc(&gpu_ptr, size_bytes);

  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  auto ptr = static_cast<Tv*>(vec_info.ptr);
  error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  run_kernel<Tv>(gpu_ptr, sca, size);

  error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  error = hipFree(gpu_ptr);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

PYBIND11_MODULE(basic_module, m)
{
  m.def("multiply_with_scalar", &map_array<double>);
  m.def("multiply_with_scalar", &map_array<float>);
}
